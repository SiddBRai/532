
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <pthread.h>
#include <assert.h>

#include <vector>
#include <unordered_map>
#include <iostream>
#include <fstream>
#include <numeric>

//#include "lock.h"

using namespace std;

typedef struct trans_node {
	int value;
} TransNode;

typedef struct {
	int trans_no;
    int item_size;
    int item_code[1024];
} Transaction;

typedef struct {
	int item_no;
    int freq;
    int trans_array_size;
    int trans_array[256];
} Item;

typedef struct {
    int freq;
    int item_set_size;
	int item_set_code[256];
    int trans_array_size;
    int trans_array[256];

    /* the indices of previous sets */
    int set1_index;
    int set2_index;

} ItemSet;


#define TRANS_NUM 600000
#define NUM_THREADS 16


__global__ 
void item_freq_count(int num_trans, Transaction *transArray, Item* itemArray)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = gridDim.x*blockDim.x;
    int i = tid;
    while ( i < num_trans) {
        int item_size = transArray[i].item_size;                 
        for (int j = 0; j < item_size; j++) {
            int item_code = transArray[i].item_code[j];
            //itemArray[item_code].freq++;
            atomicAdd(&(itemArray[item_code].freq), 1);
            /* push the transaction to the item struct */
            int _idx = atomicAdd(&(itemArray[item_code].trans_array_size), 1);
            itemArray[item_code].trans_array[_idx] = i;
        }
        i += num_threads;
    }
}

__global__
void select_with_min_support(int num_items, Item* itemArray, int min_support, ItemSet* itemsetArray, int* globalIdx)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = gridDim.x * blockDim.x;
    int i = tid;
    while (i < num_items) { 
        if (itemArray[i].freq >= min_support) {
            /* get a place in itemsetArray */
            int _idx = atomicAdd(globalIdx, 1);
            itemsetArray[_idx].freq = itemArray[i].freq;
            itemsetArray[_idx].item_set_size = 1;
            itemsetArray[_idx].item_set_code[0] = itemArray[i].item_no;
            itemsetArray[_idx].trans_array_size = itemArray[i].trans_array_size;
            memcpy(itemsetArray[_idx].trans_array, itemArray[i].trans_array, itemArray[i].trans_array_size*sizeof(int));
        }
        i += num_threads;
    }
}


__device__
bool alreadyHasTrans(ItemSet* _item_set, int trans_no)
{
    bool has = false;
    for (int i = 0; i < _item_set->trans_array_size; i++ ) {
        if (_item_set->trans_array[i] == trans_no) {
            has = true;
        }
    }
    return has;
}

/* search for transactions in the previous itemset, updating the transaction records
    and returning the count */
__device__
int find_support_count_for_itemset(ItemSet* candidate_itemset, ItemSet* checked_itemset, Transaction* trans_array)
{
    int count = 0;
    for (int i = 0; i < checked_itemset->trans_array_size; i++) {
        int trans_idx = checked_itemset->trans_array[i];
        Transaction* trans = &(trans_array[trans_idx]);
        bool itemset_found = true;
        int trans_no = -1;
        for (int j = 0; j < candidate_itemset->item_set_size; j++) {
            int target_item_code = candidate_itemset->item_set_code[j];
            bool single_item_found = false;
            for (int k = 0; k < trans->item_size; k++) {
                if (target_item_code == trans->item_code[k] && 
                    !alreadyHasTrans(candidate_itemset, trans_idx)) {
                    single_item_found = true; 
                    trans_no = trans_idx;
                    break;
                }
            }
            itemset_found &= single_item_found;
        }
        if (itemset_found) {
            candidate_itemset->trans_array[candidate_itemset->trans_array_size++] = trans_no;
            count++;
        }
    }
    return count;
}

__global__
void find_support_count(int* candidateSetSize, ItemSet* candidateSet, int* globalIdx, ItemSet* currSet, Transaction* trans_array)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int num_threads = gridDim.x * blockDim.x;
    int i = tid;
    
    while (i < *candidateSetSize) {
        int set1_idx = candidateSet[i].set1_index;
        int set2_idx = candidateSet[i].set2_index;
        int count1 = find_support_count_for_itemset(&(candidateSet[i]), &(currSet[set1_idx]), trans_array);
        int count2 = find_support_count_for_itemset(&(candidateSet[i]), &(currSet[set2_idx]), trans_array);
        
        int _global_idx = atomicAdd(globalIdx, 1);
        candidateSet[_global_idx].freq = count1 + count2;

        i += num_threads;
    }

}

int itemsetComp(const void* a, const void* b)
{
    ItemSet* set_a = (ItemSet*)(a);
    ItemSet* set_b = (ItemSet*)(b);
    int size = set_a->item_set_size;
    for (int i = 0; i < size; i++) {
        if (set_a->item_set_code[i] > set_b->item_set_code[i]) {
            return 1;
        }
        else if (set_a->item_set_code[i] < set_b->item_set_code[i]) {
            return -1;
        }
    }
    return 0;
}


int find_last_eq_class_item(int array_size, ItemSet* itemset_array, int base_pos, int start_pos, int cardinality)
{
    ItemSet* base_item_set = &(itemset_array[base_pos]);
    int last_pos = -1;
    
    if (cardinality < 2) {
        return -1;
    }

    for (int i = start_pos; i < array_size; i++) {
        ItemSet* check_item_set = &(itemset_array[i]);
        for (int j = 0; j < cardinality-1; j++) {
            if (base_item_set->item_set_code[j] != check_item_set->item_set_code[j]) {
                break; 
            }
        }
        last_pos = i;
    }

    return last_pos;
}

void* genNextItemSetArray(int itemset_array_size, ItemSet* curr_itemset_array, int nextCardinality, int* nextSize)
{
    int _arr_size = itemset_array_size;
    int new_idx = 0;
    if (itemset_array_size <= 0) {
        return NULL;
    }
    
    assert(nextCardinality == curr_itemset_array[0].item_set_size);
    
    ItemSet* next_set = NULL;
    
    if (nextCardinality == 2) {
        int next_size = (_arr_size*(_arr_size-1)) / 2;
        next_set = (ItemSet*)malloc(next_size*sizeof(ItemSet));
        memset(next_set, 0, next_size*sizeof(ItemSet));
        for (int i = 0; i < _arr_size-1; i++) {
            for (int j = i+1; j < _arr_size; j++) {
                /* set up new itemset */
                next_set[new_idx].item_set_size = nextCardinality;
                next_set[new_idx].item_set_code[0] = curr_itemset_array[i].item_set_code[0];
                next_set[new_idx].item_set_code[1] = curr_itemset_array[j].item_set_code[0];
                
                /* store the indices */
                next_set[new_idx].set1_index = i;
                next_set[new_idx].set2_index = j;

                new_idx++;
            }
        }
        *nextSize = next_size;
    }
    else {
        int i = 0;
        vector< pair<int,int> > ranges_vec;
        while (i < itemset_array_size) {
            int j = find_last_eq_class_item(itemset_array_size, curr_itemset_array, i, i+1, nextCardinality-1);
            ranges_vec.push_back(make_pair(i,j));
            i = j+1; 
        }
       
        auto pairSum = [](vector< pair<int,int> >& _vec) {
            int sum = 0;
            for (int i = 0; i < _vec.size(); i++) {
                sum += (_vec[i].second-_vec[i].first+1);
            }
            return sum;
        };
        /* allocate next level item set memory */ 
        int next_size = pairSum(ranges_vec);
        next_set = (ItemSet*)malloc(next_size*sizeof(ItemSet));
        memset(next_set, 0, next_size*sizeof(ItemSet));
        for (auto range : ranges_vec) {
            /* the priori nextCardinality-2 items should be the same */
            for (int start_pos = range.first; start_pos <= range.second-1; start_pos++) {
                for (int end_pos = start_pos+1; end_pos <= range.second; end_pos++) {
                    /* set up new itemset */
                    next_set[new_idx].item_set_size = nextCardinality;
                    
                    memcpy(next_set[new_idx].item_set_code,
                           curr_itemset_array[start_pos].item_set_code,
                           curr_itemset_array[start_pos].item_set_size*sizeof(int));
                    
                    next_set[new_idx].item_set_code[nextCardinality-1] = curr_itemset_array[end_pos].item_set_code[nextCardinality-2];
                    
                    /* store the indices */
                    next_set[new_idx].set1_index = start_pos;
                    next_set[new_idx].set2_index = end_pos;
    
                    new_idx++; 
                }
            }
        }
        *nextSize = next_size;
    }

    return (void*)next_set;
}

int main(void) 
{
    fstream fs;
    string line;
    unordered_map<string, int> item_code_map;
    unordered_map<int, int> transaction_map;

    int trans_count = 0;    /* number of transactions */
    int item_count = 0;     /* number of unique items */
    int min_support = 5;    /* mininum supoort of items */

    Transaction *transArray = (Transaction*)malloc(TRANS_NUM*sizeof(Transaction));
    memset(transArray, 0, TRANS_NUM*sizeof(Transaction));

    /* read from the file */
    fs.open("ex_data.csv", ios::in);
    while (getline(fs, line)) {
        /* get transaction number */
        ssize_t pos = line.find(",");
        int trans_no = atoi(line.substr(0, pos).c_str());
        ssize_t pos2 = line.find(",", pos+1);
        string item = line.substr(pos+1, pos2-pos-1);

        /* find item number */
        if (item_code_map.find(item) == item_code_map.end()) {
            item_code_map[item] = item_count++;
        }
        /* find transaction number */
        if (transaction_map.find(trans_no) == transaction_map.end()) {
            transArray[trans_count].trans_no = trans_count;
            transArray[trans_count].item_code[transArray[trans_count].item_size++] = item_code_map[item];
            transaction_map[trans_no] = trans_count;
            trans_count++;
        }
        else {
            int _idx = transaction_map[trans_no]; 
            transArray[_idx].item_code[transArray[_idx].item_size++] = item_code_map[item];
        }
    }
    fs.close();
    
    Item *itemArray = (Item*)malloc(item_count*sizeof(Item));
    memset(itemArray, 0, item_count*sizeof(Item));
    for (int i = 0; i < item_count; i++) {
        itemArray[i].item_no = i;
    }
    
    /* request cuda memory */
    Transaction *dev_transArray = NULL;
    hipMalloc(&dev_transArray, TRANS_NUM*sizeof(Transaction));
    hipMemcpy(dev_transArray, transArray, TRANS_NUM*sizeof(Transaction), hipMemcpyHostToDevice);
    
    Item *dev_itemArray = NULL;
    hipMalloc(&dev_itemArray, item_count*sizeof(Item));
    hipMemcpy(dev_itemArray, itemArray, item_count*sizeof(Item), hipMemcpyHostToDevice);
    
    /* calculate single item frequency */
    dim3 gridSize(256);
    dim3 blockSize(16);
    item_freq_count<<<gridSize, blockSize>>>(trans_count, dev_transArray, dev_itemArray);

    /* copy the results back to host */
    hipMemcpy(itemArray, dev_itemArray, item_count*sizeof(Item), hipMemcpyDeviceToHost);

    /* start to prune */
    int globalIdx = 0;
    int *dev_globalIdx = NULL;
    hipMalloc(&dev_globalIdx, sizeof(int));
    hipMemcpy(dev_globalIdx, &globalIdx, sizeof(int), hipMemcpyHostToDevice);

    ItemSet *itemsetArray = (ItemSet*)malloc(item_count*sizeof(ItemSet));
    memset(itemsetArray, 0, item_count*sizeof(ItemSet));
    
    ItemSet *dev_itemsetArray = NULL;
    hipMalloc(&dev_itemsetArray, item_count*sizeof(ItemSet));
    hipMemcpy(dev_itemsetArray, itemsetArray, item_count*sizeof(ItemSet), hipMemcpyHostToDevice);
    
    /* kernel doing selection for single item with minimum support */
    select_with_min_support<<<gridSize, blockSize>>>(item_count, dev_itemArray, min_support, dev_itemsetArray, dev_globalIdx);

    hipMemcpy(itemsetArray, dev_itemsetArray, item_count*sizeof(ItemSet), hipMemcpyDeviceToHost);
    hipMemcpy(&globalIdx, dev_globalIdx, sizeof(int), hipMemcpyDeviceToHost);
    
    /* Now we get the transposed database that every item set with size 1 has a corresponding list of transactions */
    /* Generate itemset with size 2 */
    
    int cardinality = 2;
    int currSetSize = item_count;
    int candidateSetSize = 0;
    int *dev_candidateSetSize = NULL;
    ItemSet* currSet = itemsetArray;
    ItemSet* dev_currSet = NULL;
    ItemSet* candidateSet = NULL;
    ItemSet* dev_candidateSet = NULL;

    hipMalloc(&dev_candidateSetSize, sizeof(int));

    while (true) {
        candidateSet = (ItemSet*)genNextItemSetArray(currSetSize, currSet, cardinality, &candidateSetSize);
        if (candidateSetSize == 0) {
            break;
        }
        assert(candidateSet != NULL);          
        
        /* allocate GPU kernel memory */
        hipMemcpy(dev_candidateSetSize, &candidateSetSize, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_globalIdx, &globalIdx, sizeof(int), hipMemcpyHostToDevice);
        //cudaMalloc(&dev_currSet, currSetSize*sizeof(ItemSet));
        //cudaMemcpy(dev_currSet, currSet, currSetSize*sizeof(ItemSet), cudaMemcpyHostToDevice);
        hipMalloc(&dev_candidateSet, candidateSetSize*sizeof(ItemSet));
        hipMemcpy(dev_candidateSet, candidateSet, candidateSetSize*sizeof(ItemSet), hipMemcpyHostToDevice);        
        
        /* launch the kernel */
        find_support_count<<<gridSize, blockSize>>>(dev_candidateSetSize, 
                                                     dev_candidateSet, 
                                                     dev_globalIdx, 
                                                     dev_currSet, 
                                                     dev_transArray);

        /* copy the result back */
        hipMemcpy(candidateSet, dev_candidateSet, candidateSetSize*sizeof(ItemSet), hipMemcpyDeviceToHost);

        /* update the parameters and free previously used memory */
        free(currSet);
        hipFree(dev_currSet);
        cardinality++;
        currSet = candidateSet;
        currSetSize = candidateSetSize;
        dev_currSet = dev_candidateSet; 
        globalIdx = 0;
    }
    
    /* Finally Generate association rules */

    return 0;
}
